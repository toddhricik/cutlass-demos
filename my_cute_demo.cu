#include "hip/hip_runtime.h"
#include <iostream>
#include <cutlass/cutlass.h>
#include <cutlass/numeric_types.h>
#include <cutlass/core_io.h>

// Define a simple kernel to do a little work on a cuda device
__global__ void kernel(cutlass::half_t x)
{
        // Print out the value of x * 2.0_hf
        printf("Device: %f\n", float(x * 2.0_hf));
        // Pause
        for(int i=0; i<100000; i++) {  }
}

// Main entrypoint
int main()
{
        // Declare and instantiate some cutlass type half_t variables
        cutlass::half_t x = 0.5_hf;
        cutlass::half_t y = 0.0_hf;
        // Read in a value from stdin and save the input to x
        std::cin >> x;
        // Write the Host side data for x to stdout
        std::cout << "Host: " << 2.0_hf * x << std::endl;
        // Hop into a device kernel parameterized by x
        kernel<<< dim3(1,1,1), dim3(1,1,1) >>> (x);
        // Synchronize to allow printf inside kernel function to access the propper stream
        hipDeviceSynchronize();
        // Return
        return 0;
}
