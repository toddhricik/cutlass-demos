#include "hip/hip_runtime.h"
#include <iostream>
#include <cutlass/cutlass.h>
//#include "cutlass/numeric/numeric_types.hpp"
#include <cutlass/numeric_types.h>
#include <cutlass/core_io.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cute/tensor.hpp>
#include <cute/layout.hpp>
#include <cute/util/print.hpp>

// Define a simple kernel to do a little work on a cuda device
__global__ void doSomething(cutlass::half_t x)
{
	// Print out the value of x * 2.0_hf
	printf("Device: %f\n", float(x * 2.0_hf));
	// Pause
	//for(int i=0; i < 1000000000; i++) { }
	//__syncthreads();
}

// Main entrypoint
int main()
{
	using namespace cute;

	Layout s8 = make_layout(Int<8>{});
	cute::print(s8);
	std::cout << std::endl;
	Layout d8 = make_layout(8);

	Layout s2xs4 = make_layout(make_shape(Int<2>{},Int<4>{}));
	Layout s2xd4 = make_layout(make_shape(Int<2>{},4));

	Layout s2xd4_a = make_layout(make_shape (Int< 2>{},4), make_stride(Int<12>{},Int<1>{}));
	Layout s2xd4_col = make_layout(make_shape(Int<2>{},4), LayoutLeft{});
	Layout s2xd4_row = make_layout(make_shape(Int<2>{},4), LayoutRight{});

	Layout s2xh4 = make_layout(make_shape (2,make_shape (2,2)), make_stride(4,make_stride(2,1)));
    std::cout << std::endl;
	cute::print(d8);
	std::cout << std::endl;
	cute::print(s2xs4);
	std::cout << std::endl;
	//Return
    return 0;
}
