#include "hip/hip_runtime.h"
#include <iostream>
#include <cutlass/cutlass.h>
//#include "cutlass/numeric/numeric_types.hpp"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <cute/tensor.hpp>
#include <cute/layout.hpp>
#include <cute/util/print.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

template <class TensorS, class TensorD, class ThreadLayoutS, class ThreadLayoutD>
__global__ static void __launch_bounds__(256, 1)
transposeKernelNaive(TensorS const S, TensorD const DT,
                ThreadLayoutS const tS, ThreadLayoutD const tD) {
  using namespace cute;
  using Element = typename TensorS::value_type;

  Tensor gS = S(make_coord(_, _), blockIdx.x, blockIdx.y);   // (bM, bN)
  Tensor gDT = DT(make_coord(_, _), blockIdx.x, blockIdx.y); // (bN, bM)

  Tensor tSgS = local_partition(gS, tS, threadIdx.x); // (ThrValM, ThrValN)
  Tensor tDgDT = local_partition(gDT, tD, threadIdx.x);

  Tensor rmem = make_tensor_like(tSgS);

  copy(tSgS, rmem);
  copy(rmem, tDgDT);
}


// Main entrypoint
int main()
{
	using namespace cute;
	// This section is for construction of tensors
	int M = 2048, N = 2048;
	float *d_S, *d_D;
	// Allocate and initialize d_S and d_D on device (omitted).
	
	// Create the row major layouts.
	auto tensor_shape = make_shape(M, N);
	auto tensor_shape_trans = make_shape(N, M);
	auto gmemLayoutS = make_layout(tensor_shape, GenRowMajor{});
	auto gmemLayoutD = make_layout(tensor_shape_trans, GenRowMajor{});

	// Create the row major tensors.
	Tensor tensor_S = make_tensor(make_gmem_ptr(d_S), gmemLayoutS);
	Tensor tensor_D = make_tensor(make_gmem_ptr(d_D), gmemLayoutD);

	// Create a column major layout. Note that we use (M,N) for shape.
	auto gmemLayoutDT = make_layout(tensor_shape, GenColMajor{});

	// Create a column major view of the dst tensor.
	Tensor tensor_DT = make_tensor(make_gmem_ptr(d_D), gmemLayoutDT);
	// This section is for tiling tensors
	using bM = Int<64>;
	using bN = Int<64>;
 
	auto block_shape = make_shape(bM{}, bN{});       // (bM, bN)
	auto block_shape_trans = make_shape(bN{}, bM{}); // (bN, bM)

	Tensor tiled_tensor_S = tiled_divide(tensor_S, block_shape); // ((bM, bN), m', n')
	Tensor tiled_tensor_DT = tiled_divide(tensor_DT, block_shape_trans); // ((bN, bM), n', m')

	auto threadLayoutS =
		make_layout(make_shape(Int<8>{}, Int<32>{}), LayoutRight{});
	auto threadLayoutD =
		make_layout(make_shape(Int<8>{}, Int<32>{}), LayoutRight{});

	dim3 gridDim(
		size<1>(tiled_tensor_S),
		size<2>(tiled_tensor_S)); // Grid shape corresponds to modes m' and n'
	dim3 blockDim(size(threadLayoutS)); // 256 threads
	kernelTransposeNaive<<<gridDim, blockDim>>>(tiled_tensor_S, tiled_tensor_DT, threadLayoutS, threadLayoutD);
	// Return
    return 0;
}
