#include "hip/hip_runtime.h"
#include <iostream>
#include <cutlass/cutlass.h>
//#include "cutlass/numeric/numeric_types.hpp"
#include <cutlass/numeric_types.h>
#include <cutlass/core_io.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cute/tensor.hpp>
#include <cute/layout.hpp>
#include <cute/util/print.hpp>

// Define a simple kernel to do a little work on a cuda device
__global__ void doSomething(cutlass::half_t x)
{
        // Print out the value of x * 2.0_hf
        printf("Device: %f\n", float(x * 2.0_hf));
        // Pause
	//for(int i=0; i < 1000000000; i++) { }
	//__syncthreads();
}
/*
template <class Shape, class Stride>
void print2D(Layout<Shape,Stride> const& layout)
{
	for (int m = 0; m < size<0>(layout); ++m) {
		for (int n = 0; n < size<1>(layout); ++n) {
			cute::print(layout(m,n));
		}
		printf("\n");
	}
}
*/
// Main entrypoint
int main()
{
        // Declare and instantiate some cutlass type half_t variables
        cutlass::half_t x = 0.5_hf;
        cutlass::half_t y = 0.0_hf;

//	hipMalloc(void** 
        // Read in a value from stdin and save the input to x
        std::cin >> x;
        // Write the Host side data for x to stdout
        std::cout << "Host: " << 2.0_hf * x << std::endl;
        // Hop into a device kernel parameterized by x
        doSomething<<< dim3(1,1,1), dim3(1,1,1) >>>(x);
	hipDeviceSynchronize();
	for(int i=0; i<10000000; i++)
	{
		// pass
	}
        // Synchronize to allow printf inside kernel function to access the propper stream
//	if(cudaErr != hipSuccess){
//		std::cout << "problem13 " << cudaErr << std::endl;
//	}

	using namespace cute;
	int M = 2048, N = 2048;
	float *d_S, *d_D;
	// Allocate and initialize d_S and d_D on device (omitted).
	// Create the row major layouts.
	auto tensor_shape = make_shape(M, N);
	auto tensor_shape_trans = make_shape(N, M);
	auto gmemLayoutS = make_layout(tensor_shape, GenRowMajor{});
	auto gmemLayoutD = make_layout(tensor_shape_trans, GenRowMajor{});

	// Create the row major tensors.
	Tensor tensor_S = make_tensor(make_gmem_ptr(d_S), gmemLayoutS);
	Tensor tensor_D = make_tensor(make_gmem_ptr(d_D), gmemLayoutD);

	// Create a column major layout. Note that we use (M,N) for shape.
	auto gmemLayoutDT = make_layout(tensor_shape, GenColMajor{});

	// Create a column major view of the dst tensor.
	Tensor tensor_DT = make_tensor(make_gmem_ptr(d_D), gmemLayoutDT);

	Layout s8 = make_layout(Int<8>{});
	cute::print(s8);
	std::cout << std::endl;
	Layout d8 = make_layout(8);

	Layout s2xs4 = make_layout(make_shape(Int<2>{},Int<4>{}));
	Layout s2xd4 = make_layout(make_shape(Int<2>{},4));

	Layout s2xd4_a = make_layout(make_shape (Int< 2>{},4), make_stride(Int<12>{},Int<1>{}));
	Layout s2xd4_col = make_layout(make_shape(Int<2>{},4), LayoutLeft{});
	Layout s2xd4_row = make_layout(make_shape(Int<2>{},4), LayoutRight{});

	Layout s2xh4 = make_layout(make_shape (2,make_shape (2,2)), make_stride(4,make_stride(2,1)));
        std::cout << std::endl;
	cute::print(d8);
	std::cout << std::endl;
	cute::print(s2xs4);
	std::cout << std::endl;
	// Return
        return 0;
}
