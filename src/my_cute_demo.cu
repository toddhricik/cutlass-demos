#include "hip/hip_runtime.h"
#include <iostream>
#include <cutlass/cutlass.h>
//#include "cutlass/numeric/numeric_types.hpp"
#include <cutlass/numeric_types.h>
#include <cutlass/core_io.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// Define a simple kernel to do a little work on a cuda device
__global__ void doSomething(cutlass::half_t x)
{
        // Print out the value of x * 2.0_hf
        printf("Device: %f\n", float(x * 2.0_hf));
        // Pause
	//for(int i=0; i < 1000000000; i++) { }
	//__syncthreads();
}

// Main entrypoint
int main()
{
        // Declare and instantiate some cutlass type half_t variables
        cutlass::half_t x = 0.5_hf;
        cutlass::half_t y = 0.0_hf;

//	hipMalloc(void** 
        // Read in a value from stdin and save the input to x
        std::cin >> x;
        // Write the Host side data for x to stdout
        std::cout << "Host: " << 2.0_hf * x << std::endl;
        // Hop into a device kernel parameterized by x
        doSomething<<< dim3(1,1,1), dim3(1,1,1) >>>(x);
	hipDeviceSynchronize();
	for(int i=0; i<10000000; i++)
	{
		// pass
	}
        // Synchronize to allow printf inside kernel function to access the propper stream
//	if(cudaErr != hipSuccess){
//		std::cout << "problem13 " << cudaErr << std::endl;
//	}
	// Return
        return 0;
}
